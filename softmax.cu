#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>

// Thrust Library for C++ CUDA code
#include <thrust/count.h>
#include <thrust/random.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/complex.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <hip/hip_runtime.h>

#define NUM_CLASSES 10


__global__ void softmax(float* softmax_layer, int NUM_THREADS)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if ( index < NUM_THREADS )
	{

		float max = softmax_layer[index*NUM_CLASSES + 0];
		for (int i = 1; i < NUM_CLASSES; i++)
		{
			if (softmax_layer[index*NUM_CLASSES + i] > max)
			{
				max = softmax_layer[index*NUM_CLASSES + i];
			}
		}

		float sum = 0.0f;
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			// Subtract max from every element in vector and write it to softmax_layer output
			softmax_layer[index*NUM_CLASSES + i] -= max;
			// Cuda fast e^x function (expf)
			softmax_layer[index*NUM_CLASSES + i] = __expf(softmax_layer[index*NUM_CLASSES + i]); 
			// Accumulate sum of e^x_i's
			sum += softmax_layer[index*NUM_CLASSES + i];
		}
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			// Divide every element by sum
			softmax_layer[index*NUM_CLASSES + i] /= sum;
		}
		printf("Thread: %d \t Sum: %f \t Max: %f\n", index, sum, max);

	}
}

void print_results(float* pd_softmax, int NUM_THREADS)
{
	
}

int main()
{
	int NUM_THREADS = 1;
	thrust::device_vector<float> device_softmax_layer(NUM_THREADS*NUM_CLASSES);
	// Fill with 0 ... 9
	thrust::sequence(device_softmax_layer.begin(), device_softmax_layer.end());
	float* pd_softmax = thrust::raw_pointer_cast(device_softmax_layer.data());

	std::cout << "NUM THREADS: " << NUM_THREADS << std::endl;
	// Before
	for (int t = 0; t < NUM_THREADS; t++)
	{
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			std::cout << device_softmax_layer[t*NUM_CLASSES + i] << " ";
		}
		std::cout << std::endl;
	}

	softmax<<<NUM_THREADS, 1>>>(pd_softmax, NUM_THREADS);

	// After
	for (int t = 0; t < NUM_THREADS; t++)
	{
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			std::cout << device_softmax_layer[t*NUM_CLASSES + i] << " ";
		}
		std::cout << std::endl;
	}

	thrust::fill(device_softmax_layer.begin(), device_softmax_layer.end(), 9);

	std::cout << "NUM THREADS: " << NUM_THREADS << std::endl;
	// Before
	for (int t = 0; t < NUM_THREADS; t++)
	{
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			std::cout << device_softmax_layer[t*NUM_CLASSES + i] << " ";
		}
		std::cout << std::endl;
	}

	softmax<<<NUM_THREADS, 1>>>(pd_softmax, NUM_THREADS);

	// After
	for (int t = 0; t < NUM_THREADS; t++)
	{
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			std::cout << device_softmax_layer[t*NUM_CLASSES + i] << " ";
		}
		std::cout << std::endl;
	}

	thrust::sequence(device_softmax_layer.begin(), device_softmax_layer.end(), 0, 2);

	std::cout << "NUM THREADS: " << NUM_THREADS << std::endl;
	// Before
	for (int t = 0; t < NUM_THREADS; t++)
	{
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			std::cout << device_softmax_layer[t*NUM_CLASSES + i] << " ";
		}
		std::cout << std::endl;
	}

	softmax<<<NUM_THREADS, 1>>>(pd_softmax, NUM_THREADS);

	// After
	for (int t = 0; t < NUM_THREADS; t++)
	{
		for (int i = 0; i < NUM_CLASSES; i++)
		{
			std::cout << device_softmax_layer[t*NUM_CLASSES + i] << " ";
		}
		std::cout << std::endl;
	}

}
